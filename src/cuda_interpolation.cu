#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <Eigen/Dense>
#include "cuda_interpolation.cuh"

__global__ void compute_barycentric_coordinates_kernel(float* d_bary_coords, const float* d_points, const float* d_rhs) {
    int idx = threadIdx.x;
    __shared__ float T[16];
    __shared__ float rhs[4];

    if (idx < 4) {
        rhs[idx] = d_rhs[idx];
        for (int i = 0; i < 3; ++i) {
            T[idx * 4 + i] = d_points[idx * 3 + i];
        }
        T[idx * 4 + 3] = 1.0f;
    }

    __syncthreads();

    if (idx < 4) {
        float sum = 0.0f;
        for (int i = 0; i < 4; ++i) {
            sum += T[idx * 4 + i] * rhs[i];
        }
        d_bary_coords[idx] = sum;
    }
}

extern "C" void compute_barycentric_coordinates_cuda(Eigen::Vector4f& result, const float* points, const float* rhs) {
    float h_bary_coords[4];

    float *d_points, *d_rhs, *d_bary_coords;
    hipMalloc((void**)&d_points, 12 * sizeof(float));
    hipMalloc((void**)&d_rhs, 4 * sizeof(float));
    hipMalloc((void**)&d_bary_coords, 4 * sizeof(float));

    hipMemcpy(d_points, points, 12 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rhs, rhs, 4 * sizeof(float), hipMemcpyHostToDevice);

    compute_barycentric_coordinates_kernel<<<1, 4>>>(d_bary_coords, d_points, d_rhs);

    hipMemcpy(h_bary_coords, d_bary_coords, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_rhs);
    hipFree(d_bary_coords);

    for (int i = 0; i < 4; ++i) {
        result[i] = h_bary_coords[i];
    }
}
